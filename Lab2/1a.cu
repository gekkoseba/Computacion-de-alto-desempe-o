#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


struct AoS{
	int up;
	int left;
	int right;
	int down;
};

struct SoA{
	int* up;
	int* left;
	int* right;
	int* down;
};

void printAoS(struct AoS *array,int size){
	for (int i = 0; i < size; ++i)
	{
		printf("%d ",array[i].up );
		printf("%d ",array[i].left );
		printf("%d ",array[i].right );
		printf("%d\n",array[i].down );
	}
}
void checkMassAoS(struct AoS *array,int size){
	int sum=0;
	for (int i = 0; i < size; ++i)
	{
		sum+=array[i].up +array[i].left+array[i].right+array[i].down;
	}
	printf("%d\n",sum );
}
__global__ void collision_kernel_AoS(struct AoS *array,int size){
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	if (myID<size)
	{
		if (array[myID].up==1 && array[myID].down==1 && array[myID].right==0 && array[myID].left==0)
		{
			array[myID].up=0;
			array[myID].down=0;
			array[myID].left=1;
			array[myID].right=1;
		}
		else if (array[myID].up==0 && array[myID].down==0 && array[myID].right==1 && array[myID].left==1)
		{
			array[myID].up=1;
			array[myID].down=1;
			array[myID].left=0;
			array[myID].right=0;
		}
	}
}
void printSoA(struct SoA structure,int size){
	for (int i = 0; i < size; ++i)
	{
		printf("%d ",structure.up[i] );
		printf("%d ",structure.left[i] );
		printf("%d ",structure.right[i] );
		printf("%d\n",structure.down[i] );
	}
}
void checkMassSoA(struct SoA structure,int size){
	int sum=0;
	for (int i = 0; i < size; ++i)
	{
		sum+=structure.up[i]+structure.left[i] +structure.right[i] +structure.down[i];
	}
	printf("%d\n",sum );
}
__global__ void collision_kernel_SoA(struct SoA structure,int size){
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	if (myID<size)
	{
		if (structure.up[myID]==1 && structure.down[myID]==1 && structure.right[myID]==0 && structure.left[myID]==0)
		{
			structure.up[myID]=0;
			structure.down[myID]=0;
			structure.left[myID]=1;
			structure.right[myID]=1;
		}
		else if (structure.up[myID]==0 && structure.down[myID]==0 && structure.right[myID]==1 && structure.left[myID]==1)
		{
			structure.up[myID]=1;
			structure.down[myID]=1;
			structure.left[myID]=0;
			structure.right[myID]=0;
		}
	}
}
void initSoA(){
	FILE* file = fopen ("initial.txt", "r");
  	int N = 0;
  	int M = 0;
	fscanf (file, "%d", &N);    
	fscanf (file, "%d", &M);  
	struct SoA structure;
	structure.up =(int*) malloc(sizeof(int)*N*M); 
	structure.down =(int*) malloc(sizeof(int)*N*M); 
	structure.left =(int*) malloc(sizeof(int)*N*M); 
	structure.right =(int*) malloc(sizeof(int)*N*M); 
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.right[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.up[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.left[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.down[i]);
	}
	fclose (file);   
	checkMassSoA(structure,M*N);
	return ;
}
void initAoS(){
	FILE* file = fopen ("initial.txt", "r");
  	int N = 0;
  	int M = 0;
	fscanf (file, "%d", &N);    
	fscanf (file, "%d", &M);  
	struct AoS* array =(struct AoS*) malloc(sizeof(struct AoS)*N*M);
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].right);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].up);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].left);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].down);
	}
	fclose (file);   
	checkMassAoS(array,M*N);
	return ;
}

int main()
{
	initAoS();
	initSoA();
	return 0;
}