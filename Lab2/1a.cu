#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


struct AoS{
	int up;
	int left;
	int right;
	int down;
};

struct SoA{
	int* up;
	int* left;
	int* right;
	int* down;
};

void printAoS(struct AoS *array,int size){
	for (int i = 0; i < size; ++i)
	{
		printf("%d ",array[i].up );
		printf("%d ",array[i].left );
		printf("%d ",array[i].right );
		printf("%d\n",array[i].down );
	}
}
void printSoA(struct SoA structure,int size){
	for (int i = 0; i < size; ++i)
	{
		printf("%d ",structure.up[i] );
		printf("%d ",structure.left[i] );
		printf("%d ",structure.right[i] );
		printf("%d\n",structure.down[i] );
	}
}
void initSoA(){
	FILE* file = fopen ("initial.txt", "r");
  	int N = 0;
  	int M = 0;
	fscanf (file, "%d", &N);    
	fscanf (file, "%d", &M);  
	struct SoA structure;
	structure.up =(int*) malloc(sizeof(int)*N*M); 
	structure.down =(int*) malloc(sizeof(int)*N*M); 
	structure.left =(int*) malloc(sizeof(int)*N*M); 
	structure.right =(int*) malloc(sizeof(int)*N*M); 
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.right[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.up[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.left[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.down[i]);
	}
	fclose (file);   
	printSoA(structure,5);
	return ;
}
void initAoS(){
	FILE* file = fopen ("initial.txt", "r");
  	int N = 0;
  	int M = 0;
	fscanf (file, "%d", &N);    
	fscanf (file, "%d", &M);  
	struct AoS* array =(struct AoS*) malloc(sizeof(struct AoS)*N*M);
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].right);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].up);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].left);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].down);
	}
	fclose (file);   
	printAoS(array,5);
	return ;
}

int main()
{
	initAoS();
	initSoA();
	return 0;
}