#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//Arreglo de estructuras
struct AoS{
	int up;
	int left;
	int right;
	int down;
};
//Estructura de arreglos
struct SoA{
	int* up;
	int* left;
	int* right;
	int* down;
};
//Imprime arreglo de estructuras
void printAoS(int* array,int size){
	for (int i = 0; i < size; ++i)
	{
		printf("%d ",array[i*4] );
		printf("%d ",array[i*4+1] );
		printf("%d ",array[i*4+2] );
		printf("%d\n",array[i*4+3] );
	}
}
//imprime masa del sistema de arreglo de estructuras
void checkMassAoS(int *array,int size){
	int sum=0;
	for (int i = 0; i < size; ++i)
	{
		sum+=array[i*4] +array[i*4+2]+array[i*4+3]+array[i*4+1];
	}
	printf("%d\n",sum );
}
//Kernel de colision de arreglo de estructura
__global__ void collision_kernel_AoS(int *array,int size){
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	printf("ho\n");
	if (myID<size)
	{
		printf("la\n");
		if (array[myID*4]==1 && array[myID*4+1]==1 && array[myID*4+3]==0 && array[myID*4+2]==0)
		{
			array[myID*4]=0;
			array[myID*4+1]=0;
			array[myID*4+2]=1;
			array[myID*4+3]=1;
		}
		else if (array[myID*4]==0 && array[myID*4+1]==0 && array[myID*4+3]==1 && array[myID*4+2]==1)
		{
			array[myID*4]=1;
			array[myID*4+1]=1;
			array[myID*4+2]=0;
			array[myID*4+3]=0;
		}
	}
}
//Hace todo para arreglo de estructuras
void initAoS(){
	//leer archivo
	FILE* file = fopen ("a.txt", "r");
  	int N = 0;
  	int M = 0;
	fscanf (file, "%d", &N);    
	fscanf (file, "%d", &M);  
	//crea y llena arreglo de estructuras
	int* array =(int*) malloc(sizeof(int)*N*M*4);
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i*4+3]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i*4]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i*4+2]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i*4+1]);
	}
	fclose (file);   
	printAoS(array,M*N);
	//inicia llamada a kernel de colision, ACA HAY ERROR
	int block_size = 256;
	int grid_size = (int)ceil((float)(N * M*4) / block_size);
	int* gpuArray;
	int* array2 =(int*)malloc(sizeof(int)*N*M*4);
	hipMalloc(&gpuArray, sizeof(int)*N*M*4);
	hipMemcpy(gpuArray,array,sizeof(int)*N*M*4,hipMemcpyHostToDevice);
	collision_kernel_AoS<<<1,1>>>(gpuArray,M*N);
	hipDeviceSynchronize();
	//collision_kernel_AoS<<<grid_size,block_size>>>(gpuArray,M*N);
	hipMemcpy(array2,gpuArray,sizeof(int)*N*M*4,hipMemcpyDeviceToHost);
	hipFree(gpuArray);
	printAoS(array2,M*N);
	return ;
}
//Imprime estructura de arreglo
void printSoA(struct SoA structure,int size){
	for (int i = 0; i < size; ++i)
	{
		printf("%d ",structure.right[i] );
		printf("%d ",structure.up[i] );
		printf("%d ",structure.left[i] );
		printf("%d\n",structure.down[i] );
	}
}
//imprime masa de sistema de estructura de arreglos
void checkMassSoA(struct SoA structure,int size){
	int sum=0;
	for (int i = 0; i < size; ++i)
	{
		sum+=structure.up[i]+structure.left[i] +structure.right[i] +structure.down[i];
	}
	printf("%d\n",sum );
}
//kernel de colision de estructura de arreglos
__global__ void collision_kernel_SoA(struct SoA structure,int size){
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	if (myID<size)
	{
		if (structure.up[myID]==1 && structure.down[myID]==1 && structure.right[myID]==0 && structure.left[myID]==0)
		{
			structure.up[myID]=0;
			structure.down[myID]=0;
			structure.left[myID]=1;
			structure.right[myID]=1;
		}
		else if (structure.up[myID]==0 && structure.down[myID]==0 && structure.right[myID]==1 && structure.left[myID]==1)
		{
			structure.up[myID]=1;
			structure.down[myID]=1;
			structure.left[myID]=0;
			structure.right[myID]=0;
		}
	}
}
//Hace todo para estructura de arreglos
void initSoA(){
	FILE* file = fopen ("initial.txt", "r");
  	int N = 0;
  	int M = 0;
  	//Leer archivo
	fscanf (file, "%d", &N);    
	fscanf (file, "%d", &M);  
	//Crea y llena estructura de arreglos
	struct SoA structure;
	structure.up =(int*) malloc(sizeof(int)*N*M); 
	structure.down =(int*) malloc(sizeof(int)*N*M); 
	structure.left =(int*) malloc(sizeof(int)*N*M); 
	structure.right =(int*) malloc(sizeof(int)*N*M); 
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.right[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.up[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.left[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.down[i]);
	}
	fclose (file);   
	checkMassSoA(structure,M*N);
	//AUN NO EMPIEZO LAS LLAMADAS A LOS KERNEL, PRIMERO HARE EL AOS
	return ;
}

int main()
{
	initAoS();
	return 0;
}