#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//Arreglo de estructuras
struct AoS{
	int up;
	int left;
	int right;
	int down;
};
//Estructura de arreglos
struct SoA{
	int* up;
	int* left;
	int* right;
	int* down;
};
//Imprime arreglo de estructuras
void printAoS(struct AoS *array,int size){
	for (int i = 0; i < size; ++i)
	{
		printf("%d ",array[i].right );
		printf("%d ",array[i].up );
		printf("%d ",array[i].left );
		printf("%d\n",array[i].down );
	}
}
//imprime masa del sistema de arreglo de estructuras
void checkMassAoS(struct AoS *array,int size){
	int sum=0;
	for (int i = 0; i < size; ++i)
	{
		sum+=array[i].up +array[i].left+array[i].right+array[i].down;
	}
	printf("%d\n",sum );
}
//Kernel de colision de arreglo de estructura
__global__ void collision_kernel_AoS(struct AoS *array,int size){
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	if (myID<size)
	{
		if (array[myID].up==1 && array[myID].down==1 && array[myID].right==0 && array[myID].left==0)
		{
			array[myID].up=0;
			array[myID].down=0;
			array[myID].left=1;
			array[myID].right=1;
		}
		else if (array[myID].up==0 && array[myID].down==0 && array[myID].right==1 && array[myID].left==1)
		{
			array[myID].up=1;
			array[myID].down=1;
			array[myID].left=0;
			array[myID].right=0;
		}
	}
}
//Hace todo para arreglo de estructuras
void initAoS(){
	//leer archivo
	FILE* file = fopen ("a.txt", "r");
  	int N = 0;
  	int M = 0;
	fscanf (file, "%d", &N);    
	fscanf (file, "%d", &M);  
	//crea y llena arreglo de estructuras
	struct AoS* array =(struct AoS*) malloc(sizeof(struct AoS)*N*M);
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].right);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].up);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].left);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &array[i].down);
	}
	fclose (file);   
	printAoS(array,M*N);
	//inicia llamada a kernel de colision, ACA HAY ERROR
	int block_size = 256;
	int grid_size = (int)ceil((float)(N * M) / block_size);
	struct AoS* gpuArray;
	struct AoS* array2 =(struct AoS*) malloc(sizeof(struct AoS)*N*M);
	hipMalloc(&gpuArray, sizeof(struct AoS)*N*M);
	hipMemcpy(gpuArray,array,sizeof(struct AoS)*N*M,hipMemcpyHostToDevice);
	collision_kernel_AoS<<<1,1>>>(gpuArray,M*N);
	hipDeviceSynchronize();
	//collision_kernel_AoS<<<grid_size,block_size>>>(gpuArray,M*N);
	hipMemcpy(array2,gpuArray,sizeof(struct AoS)*N*M,hipMemcpyDeviceToHost);
	hipFree(gpuArray);
	printAoS(array2,M*N);
	return ;
}
//Imprime estructura de arreglo
void printSoA(struct SoA structure,int size){
	for (int i = 0; i < size; ++i)
	{
		printf("%d ",structure.right[i] );
		printf("%d ",structure.up[i] );
		printf("%d ",structure.left[i] );
		printf("%d\n",structure.down[i] );
	}
}
//imprime masa de sistema de estructura de arreglos
void checkMassSoA(struct SoA structure,int size){
	int sum=0;
	for (int i = 0; i < size; ++i)
	{
		sum+=structure.up[i]+structure.left[i] +structure.right[i] +structure.down[i];
	}
	printf("%d\n",sum );
}
//kernel de colision de estructura de arreglos
__global__ void collision_kernel_SoA(struct SoA structure,int size){
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	if (myID<size)
	{
		if (structure.up[myID]==1 && structure.down[myID]==1 && structure.right[myID]==0 && structure.left[myID]==0)
		{
			structure.up[myID]=0;
			structure.down[myID]=0;
			structure.left[myID]=1;
			structure.right[myID]=1;
		}
		else if (structure.up[myID]==0 && structure.down[myID]==0 && structure.right[myID]==1 && structure.left[myID]==1)
		{
			structure.up[myID]=1;
			structure.down[myID]=1;
			structure.left[myID]=0;
			structure.right[myID]=0;
		}
	}
}
//Hace todo para estructura de arreglos
void initSoA(){
	FILE* file = fopen ("initial.txt", "r");
  	int N = 0;
  	int M = 0;
  	//Leer archivo
	fscanf (file, "%d", &N);    
	fscanf (file, "%d", &M);  
	//Crea y llena estructura de arreglos
	struct SoA structure;
	structure.up =(int*) malloc(sizeof(int)*N*M); 
	structure.down =(int*) malloc(sizeof(int)*N*M); 
	structure.left =(int*) malloc(sizeof(int)*N*M); 
	structure.right =(int*) malloc(sizeof(int)*N*M); 
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.right[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.up[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.left[i]);
	}
	for (int i = 0; i < N*M; ++i)
	{
		fscanf (file, "%d", &structure.down[i]);
	}
	fclose (file);   
	checkMassSoA(structure,M*N);
	//AUN NO EMPIEZO LAS LLAMADAS A LOS KERNEL, PRIMERO HARE EL AOS
	return ;
}

int main()
{
	initAoS();
	return 0;
}