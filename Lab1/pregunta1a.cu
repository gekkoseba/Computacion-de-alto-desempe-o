
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <stdio.h>

float* CPUEuler(float t_0, float y_0, float delta_t){
	int n=10/delta_t + 1;
	float *y = (float*) malloc(sizeof(float)*n);
	for (int i=0;i<n;i++){
		y[i]=y_0;
		for (int j=0;j<i;j++){
			float t_j=t_0+j*delta_t;
			y[i]+=delta_t*(9*powf(t_j,2)-4*t_j+5);
		}
	}
	return y;
}

int main(){ 

	printf("seccion 1.a\n");
	clock_t start, end;
	float *y;
	for(int i=1;i<7;i++){
		float delta_t=pow(10,-i);
		//int n=10/delta_t + 1;
		start=clock();
		y = CPUEuler(0,4,delta_t);
		end=clock();
		double cpu_time_used = 1000 * ((double) (end - start)) / CLOCKS_PER_SEC;
		printf("%f\n",cpu_time_used);
	}
	return 0; 
} 