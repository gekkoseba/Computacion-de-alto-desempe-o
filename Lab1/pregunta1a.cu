#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


void CPUEuler(float t_0, float y_0, float delta_t){
	int n=10/delta_t +1;
	float *y = (float*) malloc(sizeof(float)*n);
	y[0]=y_0;
	for (int i=0;i<n-1;i++){
		y[i+1]=y[i]+delta_t*(9*powf(i*delta_t,2)-4*(i*delta_t)+5);
		/*
		for (int j=0;j<i;j++){
			float t_j=t_0+j*delta_t;
			y[i]+=delta_t*(9*pow(t_j,2)-4*t_j+5);
		}
		*/
	}
	free(y);
	//return y;
}

int main(int argc, char const *argv[]) {
	
	printf("seccion 1.a\n");
	clock_t start, end;
	for(int i=1;i<7;i++) {
		float delta_t=powf(10,-i);
		start=clock();
		CPUEuler(0,4,delta_t);
		end=clock();
		double cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
		printf("%f\n",cpu_time_used);
	}

	return 0;
}