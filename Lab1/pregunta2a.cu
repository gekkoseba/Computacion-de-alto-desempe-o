#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void CPUEuler2(int m, float* y_i, float delta_t,float t_i ){
	for (int i=0;i<m+1;i++){
		y_i[i]=y_i[i]+delta_t*(4*t_i-y_i[i]+3+i);
	}
}


int main(int argc, char const *argv[])
{
	printf("seccion 2.a\n");
	for (int j=4;j<9;j++){
		int m=pow(10,j);
		float *y;
		y = (float*) malloc(sizeof(float)*m+1);
		for(int i=0;i<m+1;i++){
			y[i]=i;
	 	}

		clock_t start, end;
		start=clock();
		float n=pow(10,3);
		for (int i=0;i<n+1;i++){
			float t_i=i/n;
			CPUEuler2(m,y,1/n,t_i);
		}
		end=clock();
		double cpu_time_used = ((double) (end - start)) *1000 / CLOCKS_PER_SEC;
		printf("%f\n",cpu_time_used);
	}
	return 0; 
}