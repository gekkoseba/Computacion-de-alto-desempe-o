#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void GPUEuler2(float *y, float t_i, float delta,int N) {
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	if(myID < N) {
		y[myID] = y[myID] + delta * (4*t_i - y[myID]+3+myID);
	}
}

int main(int argc, char** argv) { 

	int hilos2c[] = {64,128,256,512},bloque2c[4];
	float tiempoGPU2c, t_i2c;
	float *dev_e2c, *hst_y2c;
	hipEvent_t start2c, end2c;
	printf("seccion 2.c\n");
	for (int j=8;j<9;j++){
		int m=pow(10,j);
		hst_y2c = (float*) malloc(sizeof(float)*m+1);
		hipMalloc((void**) &dev_e2c,(m+1)*sizeof(float));
		for(int i=0;i<m+1;i++){
			hst_y2c[i]=i;
	 	}
	 	printf("%f y %f\n",hst_y2c[0],hst_y2c[m] );


	 	for(int w= 0; w<4;w++){
			bloque2c[w] = ceil((float) (m+1) /hilos2c[w]);
			hipEventCreate(&start2c);
			hipEventCreate(&end2c);
			hipEventRecord(start2c,0);
			hipMemcpy(dev_e2c, hst_y2c, (m+1)*sizeof(float), hipMemcpyHostToDevice);

			float n=powf(10,3);
			for (int i=0;i<n+1;i++){
				t_i2c = i/n;
				GPUEuler2<<<bloque2c[w],hilos2c[w]>>>(dev_e2c,t_i2c,1/n,m+1);
			}
			hipEventRecord(end2c,0);
			hipEventSynchronize(end2c);
			hipEventElapsedTime(&tiempoGPU2c,start2c,end2c);
			printf("%f\n",tiempoGPU2c);
	 	}
		hipFree(dev_e2c);
		free(hst_y2c);
	}
	return 0; 
} 