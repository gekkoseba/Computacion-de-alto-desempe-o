#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void GPUEuler2(float *y, float t_i, float delta,int N) {
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	if(myID < N) {
		y[myID] = y[myID] + delta * (4*t_i - y[myID]+3+myID);
	}
}

int main(int argc, char** argv) { 

	int hilos2b = 256,bloque2b;
	float tiempoGPU2b, t_i2b;
	float *dev_e2b, *hst_y2b;
	hipEvent_t start2b, end2b;
	printf("seccion 2.b\n");
	for (int j=4;j<9;j++){
		int m=pow(10,j);
		hst_y2b = (float*) malloc(sizeof(float)*m+1);
		hipMalloc((void**) &dev_e2b,(m+1)*sizeof(float));
		bloque2b = ceil((float) (m+1) /hilos2b);
		for(int i=0;i<m+1;i++){
			hst_y2b[i]=i;
	 	}
	 	hipEventCreate(&start2b);
		hipEventCreate(&end2b);
		hipEventRecord(start2b,0);
		hipMemcpy(dev_e2b, hst_y2b, (m+1)*sizeof(float), hipMemcpyHostToDevice);

		float n=powf(10,3);
		for (int i=0;i<n+1;i++){
			t_i2b = i/n;
			GPUEuler2<<<bloque2b,hilos2b>>>(dev_e2b,t_i2b,1/n,m+1);
		}
		hipEventRecord(end2b,0);
		hipEventSynchronize(end2b);
		hipEventElapsedTime(&tiempoGPU2b,start2b,end2b);
		printf("%f\n",tiempoGPU2b);
		hipFree(dev_e2b);
		free(hst_y2b);
	}
	return 0; 
} 
