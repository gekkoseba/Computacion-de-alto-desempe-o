#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void GPuEuler(float *y, float t_0, float y_0 ,int N, float delta) {
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	if(myID == 0){
		y[myID] = y_0;
	} else if (myID <N) {
		float j_del = delta * (myID-1);
		y[myID] = delta * (9*j_del*j_del-4*j_del+5);
	}

}

int main(int argc, char const *argv[])
{
	printf("seccion 1.b\n");
	int hilos1b = 256,n1b,bloque1b;
	float delta_t1b,tiempoGPU1b;
	float *dev_e1b;
	hipEvent_t start1b, end1b;
	for(int i=1;i<5;i++) {
		delta_t1b=powf(10,-i);
		n1b=10/delta_t1b +1;
		bloque1b = ceil((float) n1b /hilos1b);
		hipEventCreate(&start1b);
		hipEventCreate(&end1b);
		hipEventRecord(start1b,0);
		hipMalloc( (void**) &dev_e1b, n1b*sizeof(float));
		GPuEuler<<<bloque1b,hilos1b>>>(dev_e1b,0,4,n1b,delta_t1b);
		hipEventRecord(end1b,0);
		hipEventSynchronize(end1b);
		hipEventElapsedTime(&tiempoGPU1b,start1b,end1b);
		hipFree(dev_e1b);
		printf("%f\n",tiempoGPU1b);
	}

	return 0;
}