#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void GPuEuler(float *y, float t_0, float y_0 ,int N, float delta) {
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	if (myID <N) {
		y[myID] = y_0;
	    for(int i=0;i<myID;i++){
	        float j_del = delta * i+ t_0;
	        y[myID] =y[myID]+ delta * (9*j_del*j_del-4*j_del+5);
	    }
	}

}

int main(int argc, char const *argv[])
{
	printf("seccion 1.b\n");
	int hilos1b = 256,n1b,bloque1b;
	float delta_t1b,tiempoGPU1b;
	float *dev_e1b,*hst_y;
	hipEvent_t start1b, end1b;
	for(int i=1;i<5;i++) {
		delta_t1b=powf(10,-i);
		n1b=10/delta_t1b +1;
		hst_y = (float*) malloc(n1b*sizeof(float));
		bloque1b = ceil((float) n1b /hilos1b);
		hipEventCreate(&start1b);
		hipEventCreate(&end1b);
		hipEventRecord(start1b,0);
		hipMalloc( (void**) &dev_e1b, n1b*sizeof(float));
		GPuEuler<<<bloque1b,hilos1b>>>(dev_e1b,0,4,n1b,delta_t1b);
		hipEventRecord(end1b,0);
		hipEventSynchronize(end1b);
		hipEventElapsedTime(&tiempoGPU1b,start1b,end1b);
		hipMemcpy(hst_y,dev_e1b,n1b*sizeof(float),hipMemcpyDeviceToHost);
		hipFree(dev_e1b);
		free(hst_y);
		hipEventDestroy(start1b);
		hipEventDestroy(end1b);
		printf("%f\n",tiempoGPU1b);
	}

	return 0;
}