#include <time.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void hybridGPuEuler(float *y, float y_0 ,int N) {
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	if (myID <N) {
		y[myID] = y[myID] + y_0;
	}

}

float* hybridCPUEuler(float t_0, float delta_t){
	int n=10/delta_t +1;
	float *s = (float*) malloc(sizeof(float)*n);
	s[0]=0;
	for (int i=0;i<n-1;i++){
		s[i+1]=s[i]+delta_t*(9*powf(i*delta_t,2)-4*(i*delta_t)+5);
	}
	return s;
}

int main(int argc, char const *argv[])
{
	printf("seccion 1.c\n");
	int hilos1c = 256,n1c,bloque1c;
	float delta_t1c,tiempoGPU1c;
	float *dev_e1c,*hst_y;
	clock_t startcpu1c, endcpu1c;
	hipEvent_t startgpu1c, endgpu1c;
	for(int i=1;i<7;i++) {
		delta_t1c=powf(10,-i);
		n1c=10/delta_t1c +1;
		startcpu1c = clock();
		hst_y = hybridCPUEuler(0,delta_t1c);
		endcpu1c = clock();
		bloque1c = ceil((float) n1c /hilos1c);
		hipEventCreate(&startgpu1c);
		hipEventCreate(&endgpu1c);
		hipEventRecord(startgpu1c,0);
		hipMalloc( (void**) &dev_e1c, n1c*sizeof(float));
		hipMemcpy(dev_e1c,hst_y,n1c*sizeof(float),hipMemcpyHostToDevice);
		hybridGPuEuler<<<bloque1c,hilos1c>>>(dev_e1c,4,n1c);
		hipEventRecord(endgpu1c,0);
		hipEventSynchronize(endgpu1c);
		hipEventElapsedTime(&tiempoGPU1c,startgpu1c,endgpu1c);
		hipMemcpy(hst_y,dev_e1c,n1c*sizeof(float),hipMemcpyDeviceToHost);
		hipFree(dev_e1c);
		free(hst_y);
		hipEventDestroy(startgpu1c);
		hipEventDestroy(endgpu1c);
		double cpu_time_used = ((double) (endcpu1c - startcpu1c)) * 1000 / CLOCKS_PER_SEC;

		printf("tiempo en CPU: %f ms, tiempo en GPU: %f ms y el tiempo total es: %f ms\n", cpu_time_used ,tiempoGPU1c,cpu_time_used+tiempoGPU1c);
	}

	return 0;
}